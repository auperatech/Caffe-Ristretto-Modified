#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/saturation_layer.hpp"

namespace caffe {

    template <typename Dtype>
__global__ void SaturationForward(const int n, const Dtype* in, Dtype* out,
    Dtype saturation) {
        float tmp;
  CUDA_KERNEL_LOOP(index, n) {
    tmp = in[index] > saturation ? saturation : in[index];
    out[index] = tmp < (-saturation) ? (-saturation) : tmp;
  }
}

template <typename Dtype>
void SaturationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SaturationForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, saturation_);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SaturationBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype saturation) {
  CUDA_KERNEL_LOOP(index, n) {
    if(in_data[index] >= (-saturation) && in_data[index] <= saturation){
        out_diff[index] = in_diff[index];
    }
    else{
        out_diff[index] = 0;
    }
  }
}

template <typename Dtype>
void SaturationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SaturationBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, saturation_);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SaturationLayer);


}  // namespace caffe